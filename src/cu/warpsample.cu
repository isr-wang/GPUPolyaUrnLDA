#include "hip/hip_runtime.h"
#include "warpsample.cuh"
#include <thrust/system/cuda/detail/cub/block/block_scan.cuh>
//#include <thrust/scan.h>
//#include <thrust/device_ptr.h>
//#include <thrust/execution_policy.h>

namespace gplda {

__global__ void compute_d_idx(uint32_t* d_len, uint32_t* d_idx, uint32_t n_docs) {
  typedef hipcub::BlockScan<int32_t, GPLDA_COMPUTE_D_IDX_BLOCKDIM> BlockScan;
  __shared__ typename BlockScan::TempStorage temp;

  if(blockIdx.x == 0) {
    int32_t thread_d;
    int32_t initial_value = 0;
    int32_t total_value;
    for(int32_t offset = 0; offset < n_docs / blockDim.x + 1; ++offset) {
      int32_t i = threadIdx.x + offset * blockDim.x;
      if(i < n_docs) {
        thread_d = d_len[i];
      } else {
        thread_d = 0;
      }

      BlockScan(temp).ExclusiveScan(thread_d, thread_d, 0, hipcub::Sum(), total_value);

      // workaround for CUB bug: apply offset manually
      __syncthreads();
      thread_d = thread_d + initial_value;
      initial_value = total_value + initial_value;

      if(i < n_docs) {
        d_idx[i] = thread_d;
      }
    }
  }
}

__global__ void warp_sample_topics(uint32_t size, uint32_t n_docs, uint32_t *z, uint32_t *w, uint32_t *d_len, uint32_t *d_idx, float** prob, float** alias, hiprandStatePhilox4_32_10_t* rng) {
//  __shared__ uint32_t m_topics[128];
//  __shared__ uint32_t m_words[128];

  // load current row of Phi into shared memory
  // load Alias table into shared memory (worth it? we may not access it at all)
  // compute
}

}
