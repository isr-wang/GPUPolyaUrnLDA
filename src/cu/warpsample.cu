#include "hip/hip_runtime.h"
#include "hashmap.cuh"
#include "warpsample.cuh"
#include <thrust/system/cuda/detail/cub/block/block_scan.cuh>
#include <thrust/system/cuda/detail/cub/warp/warp_scan.cuh>

namespace gplda {

__global__ void compute_d_idx(uint32_t* d_len, uint32_t* d_idx, uint32_t n_docs) {
  typedef hipcub::BlockScan<int32_t, GPLDA_COMPUTE_D_IDX_BLOCKDIM> BlockScan;
  __shared__ typename BlockScan::TempStorage temp;

  if(blockIdx.x == 0) {
    int32_t thread_d;
    int32_t initial_value = 0;
    int32_t total_value;
    for(int32_t offset = 0; offset < n_docs / blockDim.x + 1; ++offset) {
      int32_t i = threadIdx.x + offset * blockDim.x;
      if(i < n_docs) {
        thread_d = d_len[i];
      } else {
        thread_d = 0;
      }

      BlockScan(temp).ExclusiveScan(thread_d, thread_d, 0, hipcub::Sum(), total_value);

      // workaround for CUB bug: apply offset manually
      __syncthreads();
      thread_d = thread_d + initial_value;
      initial_value = total_value + initial_value;

      if(i < n_docs) {
        d_idx[i] = thread_d;
      }
    }
  }
}

__device__ __forceinline__ uint32_t draw_alias(float u, float* prob, uint32_t* alias, uint32_t table_size, int32_t lane_idx) {
  uint32_t ret = 0;
  if(lane_idx == 0) {
    // determine the slot and update random number
    float ts = (float) table_size;
    uint32_t slot = (uint32_t) (u * ts);
    u = fmodf(u, __frcp_rz(ts)) * ts;

    // load table elements from global memory
    float thread_prob = prob[slot];
    uint32_t thread_alias = alias[slot];

    // return the resulting draw
    if(u < thread_prob) {
      ret = slot;
    } else {
      ret = thread_alias;
    }
  }
  return ret;
}

__device__ __forceinline__ uint32_t draw_wary_search(float u) {
  return 0;
}

__device__ __forceinline__ void count_topics(uint32_t* z, uint32_t document_size, HashMap* m, void* temp, int32_t lane_idx, hiprandStatePhilox4_32_10_t* rng) {
  // initialize the hash table
  hash_map_init(m, temp, document_size, warpSize, rng);

  // loop over z, add to m
  for(int32_t offset = 0; offset < document_size / warpSize + 1; ++offset) {
    int32_t i = offset * warpSize + lane_idx;
    if(i < document_size) {
      hash_map_accumulate(z[i], i, m);
    }
  }
}

__device__ __forceinline__ float compute_product_cumsum(uint32_t* mPhi, HashMap* m, float* Phi_dense, int32_t warp_idx, hipcub::WarpScan<int32_t>::TempStorage* temp) {
  int32_t thread_mPhi = 0;
  hipcub::WarpScan<int32_t>(temp[warp_idx]).ExclusiveSum(thread_mPhi, thread_mPhi);
  return 0.0f;
}

__global__ void warp_sample_topics(uint32_t size, uint32_t n_docs,
    uint32_t* z, uint32_t* w, uint32_t* d_len, uint32_t* d_idx, uint32_t* K_d, void* temp,
    uint32_t K, uint32_t V, uint32_t max_K_d,
    float* Phi_dense,
    float** prob, uint32_t** alias, hiprandStatePhilox4_32_10_t* rng) {
  // initialize variables
  int32_t lane_idx = threadIdx.x % warpSize;
  int32_t warp_idx = threadIdx.x / warpSize;
  hiprandStatePhilox4_32_10_t warp_rng = rng[0];
  HashMap m;
  uint32_t** mPhi = (uint32_t**) &m.temp_data;
  __shared__ typename hipcub::WarpScan<int32_t>::TempStorage warp_scan_temp[1];

  // loop over documents
  for(int32_t i = 0; i < n_docs; ++i) {
    // count topics in document
    uint32_t warp_d_len = d_len[i];
    uint32_t warp_d_idx = d_idx[i];
    count_topics(z + warp_d_idx * sizeof(uint32_t), warp_d_len, &m, temp, lane_idx, &warp_rng);

    // loop over words
    for(int32_t j = 0; j < warp_d_len; ++j) {
      // load z,w from global memory
      uint32_t warp_z = z[warp_d_idx + j];
      uint32_t warp_w = 0;//w[warp_d_idx + j]; // why is this broken?

      // remove current z from sufficient statistic
      hash_map_accumulate(warp_z, lane_idx == 0 ? -1 : 0, &m); // decrement on 1st lane without branching

      // compute m*phi and sigma_b
      float warp_sigma_a = 0.0f;
      float sigma_b = compute_product_cumsum(*mPhi, &m, Phi_dense, warp_idx, warp_scan_temp);

      // update z
      float u1 = hiprand_uniform(&warp_rng);
      float u2 = hiprand_uniform(&warp_rng);
      if(u1 * (warp_sigma_a + sigma_b) > warp_sigma_a) {
        // sample from m*Phi
        warp_z = draw_wary_search(u2);
      } else {
        // sample from alias table
        warp_z = draw_alias(u2, prob[warp_w], alias[warp_w], /*table_size =*/ 1, lane_idx);
      }

      // add new z to sufficient statistic
      hash_map_accumulate(warp_z, lane_idx == 0, &m); // increment on 1st lane without branching
      if(lane_idx == 0) {
        z[warp_d_idx + j] = warp_z;
      }
    }
  }
}

}
