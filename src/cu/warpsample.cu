#include "hip/hip_runtime.h"
#include "hashmap.cuh"
#include "warpsample.cuh"
#include <thrust/system/cuda/detail/cub/block/block_scan.cuh>

namespace gplda {

__global__ void compute_d_idx(uint32_t* d_len, uint32_t* d_idx, uint32_t n_docs) {
  typedef hipcub::BlockScan<int32_t, GPLDA_COMPUTE_D_IDX_BLOCKDIM> BlockScan;
  __shared__ typename BlockScan::TempStorage temp;

  if(blockIdx.x == 0) {
    int32_t thread_d;
    int32_t initial_value = 0;
    int32_t total_value;
    for(int32_t offset = 0; offset < n_docs / blockDim.x + 1; ++offset) {
      int32_t i = threadIdx.x + offset * blockDim.x;
      if(i < n_docs) {
        thread_d = d_len[i];
      } else {
        thread_d = 0;
      }

      BlockScan(temp).ExclusiveScan(thread_d, thread_d, 0, hipcub::Sum(), total_value);

      // workaround for CUB bug: apply offset manually
      __syncthreads();
      thread_d = thread_d + initial_value;
      initial_value = total_value + initial_value;

      if(i < n_docs) {
        d_idx[i] = thread_d;
      }
    }
  }
}

__device__ __forceinline__ uint32_t draw_alias(float u, float* prob, uint32_t* alias, uint32_t table_size) {
  // determine the slot and update random number
  float ts = (float) table_size;
  uint32_t slot = (uint32_t) (u * ts);
  u = fmodf(u, __frcp_rz(ts)) * ts;

  // load table elements from global memory
  float thread_prob = prob[slot];
  uint32_t thread_alias = alias[slot];

  // return the resulting draw
  if(u < thread_prob) {
    return slot;
  } else {
    return thread_alias;
  }
}

__device__ __forceinline__ uint32_t draw_wary_search(float u) {
  return 0;
}

__device__ __forceinline__ void count_topics(uint32_t* z, uint32_t document_size, HashMap* m, void* temp) {
  hash_map_init(m, temp, document_size);
  // loop over z, add to m
}

__device__ __forceinline__ float compute_product_cumsum(uint32_t* mPhi) {
  return 0.0f;
}

__global__ void warp_sample_topics(uint32_t size, uint32_t n_docs,
    uint32_t* z, uint32_t* w, uint32_t* d_len, uint32_t* d_idx, uint32_t* K_d, void* temp,
    uint32_t K, uint32_t V, uint32_t max_K_d,
    float** prob, uint32_t** alias, hiprandStatePhilox4_32_10_t* rng) {
  // initialize variables
  hiprandStatePhilox4_32_10_t warp_rng = rng[0];
  HashMap m;
  uint32_t** mPhi = (uint32_t**) &m.temp_data;

  // loop over documents
  for(int32_t i = 0; i < n_docs; ++i) {
    // count topics in document
    uint32_t warp_d_len = d_len[i];
    uint32_t warp_d_idx = d_idx[i];
    count_topics(z + warp_d_idx * sizeof(uint32_t), warp_d_len, &m, temp);

    // loop over words
    for(int32_t j = 0; j < warp_d_len; ++j) {
      // compute m*phi and sigma_b
      uint32_t warp_z = z[warp_d_idx + j];
      uint32_t warp_w = w[warp_d_idx + j];
      float warp_sigma_a = 0.0f;
      float sigma_b = compute_product_cumsum(*mPhi);

      // update z
      float u1 = hiprand_uniform(&warp_rng);
      float u2 = hiprand_uniform(&warp_rng);
      if(u1 * (warp_sigma_a + sigma_b) > warp_sigma_a) {
        // sample from m*Phi
        z[warp_d_idx + j] = draw_wary_search(u2);
      } else {
        // sample from alias table
        z[warp_d_idx + j] = draw_alias(u2, prob[0], alias[0], /*table_size =*/ 1);
      }

    }

  }
}

}
