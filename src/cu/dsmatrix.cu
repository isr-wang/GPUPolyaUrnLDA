#include "dsmatrix.cuh"
#include "error.cuh"
#include "train.cuh"

namespace gplda {

template<class T>
DSMatrix<T>::DSMatrix<T>() {
  hipMalloc/*Pitch*/(&dense, ARGS->K * ARGS->V * sizeof(T)) >> GPLDA_CHECK;
}

template<class T>
DSMatrix<T>::~DSMatrix<T>() {
  hipFree(dense) >> GPLDA_CHECK;
}

template class DSMatrix<float>;
template class DSMatrix<uint32_t>;

}
