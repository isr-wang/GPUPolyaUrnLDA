#include "dsmatrix.cuh"
#include "error.cuh"
#include "train.cuh"

namespace gplda {

template<class T>
DSMatrix<T>::DSMatrix<T>() {
  hipMalloc/*Pitch*/(&dense, args->K * args->V * sizeof(T)) >> GPLDA_CHECK;
}

template<class T>
DSMatrix<T>::~DSMatrix<T>() {
  hipFree(dense) >> GPLDA_CHECK;
}

template class DSMatrix<float>;
template class DSMatrix<uint32_t>;

}
