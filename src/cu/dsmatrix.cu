#include "dsmatrix.h"
#include "error.h"
#include "train.h"

namespace gplda {

template<class T>
DSMatrix<T>::DSMatrix<T>() {
  hipMalloc/*Pitch*/(&dense, ARGS->K * ARGS->V * sizeof(T)) >> GPLDA_CHECK;
}

template<class T>
DSMatrix<T>::~DSMatrix<T>() {
  hipFree(dense) >> GPLDA_CHECK;
}

template class DSMatrix<float>;
template class DSMatrix<uint32_t>;

}
