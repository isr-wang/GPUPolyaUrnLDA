#include "train.h"
#include "dlhmatrix.h"
#include "poisson.h"
#include "polyaurnsampler.h"
#include "spalias.h"
#include "warpsampler.h"

#define POIS_MAX_LAMBDA 100
#define POIS_MAX_VALUE 200

namespace gplda {

Args* ARGS;
DLHMatrix* Phi;
DLHMatrix* n;
Poisson* pois;
SpAlias* alias;

extern "C" void initialize(Args* args, Buffer* buffers, size_t n_buffers) {
  ARGS = args;
  for(int i = 0; i < n_buffers; ++i) {
    hipMalloc(&buffers[i].gpu_z, buffers[i].size * sizeof(uint32_t));
    hipMalloc(&buffers[i].gpu_w, buffers[i].size * sizeof(uint32_t));
    hipMalloc(&buffers[i].gpu_d_len, buffers[i].size * sizeof(uint32_t));
    hipMalloc(&buffers[i].gpu_d_idx, buffers[i].size * sizeof(uint32_t));
  }
  Phi = new DLHMatrix();
  n = new DLHMatrix();
  pois = new Poisson(POIS_MAX_LAMBDA, POIS_MAX_VALUE);
  alias = new SpAlias();
}

extern "C" void cleanup(Buffer *buffers, size_t n_buffers) {
  delete alias;
  delete pois;
  delete n;
  delete Phi;
  for(int i = 0; i < n_buffers; ++i) {
    hipFree(buffers[i].gpu_z);
    hipFree(buffers[i].gpu_w);
    hipFree(buffers[i].gpu_d_len);
    hipFree(buffers[i].gpu_d_idx);
  }
  ARGS = NULL;
}

extern "C" void sample_phi() {
  polya_urn_sampler<<<1,1>>>();
  build_alias<<<1,1>>>();
}

extern "C" void sample_z(Buffer *buffer) {
  // copy memory
  warp_sampler<<<1,1>>>(buffer->size, buffer->n_docs, buffer->gpu_z, buffer->gpu_w, buffer->gpu_d_len, buffer->gpu_d_idx);
}

}
