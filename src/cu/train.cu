#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h> // need to add -lcurand to nvcc flags
#include <hipblas.h> // need to add -lcublas to nvcc flags
#include "assert.h"

#include "train.cuh"
#include "dsmatrix.cuh"
#include "error.cuh"
#include "poisson.cuh"
#include "polyaurn.cuh"
#include "random.cuh"
#include "spalias.cuh"
#include "tuning.cuh"
#include "warpsample.cuh"

namespace gplda {

// global variables
Args* args; // externally visible
DSMatrix<f32>* Phi;
DSMatrix<u32>* n;
Poisson* pois;
SpAlias* alias;
f32* sigma_a;
u32* C;
hiprandStatePhilox4_32_10_t* Phi_rng;
hipStream_t* Phi_stream;
hipblasHandle_t* cublas_handle;
DSMatrix<f32>* Phi_temp;
f32* d_one;
f32* d_zero;

extern "C" void initialize(Args* init_args, Buffer* buffers, u32 n_buffers) {
  // set the pointer to args struct
  args = init_args;

  // if the types are broken, explode
  assert(sizeof(u64) == 8 && sizeof(u32) == 4 && sizeof(i32) == 4 && sizeof(f32) == 4);

  // allocate and initialize cuBLAS
  cublas_handle = new hipblasHandle_t;
  hipblasCreate(cublas_handle) >> GPLDA_CHECK;
  hipblasSetPointerMode(*cublas_handle, HIPBLAS_POINTER_MODE_DEVICE) >> GPLDA_CHECK;
  f32 h_zero = 0.0f;
  hipMalloc(&d_zero, sizeof(f32)) >> GPLDA_CHECK;
  hipMemcpy(d_zero, &h_zero, sizeof(f32), hipMemcpyHostToDevice) >> GPLDA_CHECK;
  f32 h_one = 1.0f;
  hipMalloc(&d_one, sizeof(f32)) >> GPLDA_CHECK;
  hipMemcpy(d_one, &h_one, sizeof(f32), hipMemcpyHostToDevice) >> GPLDA_CHECK;
  Phi_temp = new DSMatrix<f32>();

  // allocate and initialize cuRAND
  hipMalloc(&Phi_rng, sizeof(hiprandStatePhilox4_32_10_t)) >> GPLDA_CHECK;
  rng_init<<<1,1>>>(0, 0, Phi_rng);
  hipDeviceSynchronize() >> GPLDA_CHECK;

  // allocate and initialize streams
  Phi_stream = new hipStream_t;
  hipStreamCreate(Phi_stream) >> GPLDA_CHECK;

  // allocate memory for buffers
  for(i32 i = 0; i < n_buffers; ++i) {
    buffers[i].stream = new hipStream_t;
    hipStreamCreate(buffers[i].stream) >> GPLDA_CHECK;
    hipMalloc(&buffers[i].gpu_z, args->buffer_size * sizeof(u32)) >> GPLDA_CHECK;
    hipMalloc(&buffers[i].gpu_w, args->buffer_size * sizeof(u32)) >> GPLDA_CHECK;
    hipMalloc(&buffers[i].gpu_d_len, args->max_K_d * sizeof(u32)) >> GPLDA_CHECK;
    hipMalloc(&buffers[i].gpu_d_idx, args->max_K_d * sizeof(u32)) >> GPLDA_CHECK;
    hipMalloc(&buffers[i].gpu_K_d, args->max_K_d * sizeof(u32)) >> GPLDA_CHECK;
    hipMalloc(&buffers[i].gpu_temp, 2 * (args->max_K_d + GPLDA_HASH_STASH_SIZE) * sizeof(u32)) >> GPLDA_CHECK;
    hipMalloc(&buffers[i].gpu_rng, sizeof(hiprandStatePhilox4_32_10_t)) >> GPLDA_CHECK;
    rng_init<<<1,1>>>(0, i + 1, buffers[i].gpu_rng);
    hipDeviceSynchronize() >> GPLDA_CHECK;
  }

  // allocate globals
  Phi = new DSMatrix<f32>();
  n = new DSMatrix<u32>();
  pois = new Poisson(GPLDA_POIS_MAX_LAMBDA, GPLDA_POIS_MAX_VALUE, args->beta);
  alias = new SpAlias(args->V, args->K);
  hipMalloc(&sigma_a,args->V * sizeof(f32)) >> GPLDA_CHECK;
  hipMalloc(&C,args->V * sizeof(u32)) >> GPLDA_CHECK;
  hipMemcpy(C, args->C, args->V * sizeof(u32), hipMemcpyHostToDevice) >> GPLDA_CHECK;

  // run device init code
  polya_urn_init<<<args->K,GPLDA_POLYA_URN_SAMPLE_BLOCKDIM>>>(n->dense, C, args->beta, args->V, pois->pois_alias->prob, pois->pois_alias->alias, pois->max_lambda, pois->max_value, Phi_rng);
  hipDeviceSynchronize() >> GPLDA_CHECK;
  rng_advance<<<1,1>>>(args->K*args->V,Phi_rng);
  hipDeviceSynchronize() >> GPLDA_CHECK;
}

extern "C" void cleanup(Buffer* buffers, u32 n_buffers) {
  // deallocate globals
  hipFree(C) >> GPLDA_CHECK;
  hipFree(sigma_a) >> GPLDA_CHECK;
  delete alias;
  delete pois;
  delete n;
  delete Phi;

  // deallocate memory for buffers
  for(i32 i = 0; i < n_buffers; ++i) {
    hipFree(buffers[i].gpu_z) >> GPLDA_CHECK;
    hipFree(buffers[i].gpu_w) >> GPLDA_CHECK;
    hipFree(buffers[i].gpu_d_len) >> GPLDA_CHECK;
    hipFree(buffers[i].gpu_d_idx) >> GPLDA_CHECK;
    hipFree(buffers[i].gpu_K_d) >> GPLDA_CHECK;
    hipFree(buffers[i].gpu_temp) >> GPLDA_CHECK;
    hipFree(buffers[i].gpu_rng) >> GPLDA_CHECK;
    hipStreamDestroy(*buffers[i].stream) >> GPLDA_CHECK;
    delete buffers[i].stream;
  }

  // deallocate streams
  hipStreamDestroy(*Phi_stream) >> GPLDA_CHECK;
  delete Phi_stream;

  // deallocate cuRAND
  hipFree(Phi_rng) >> GPLDA_CHECK;

  // deallocate cuBLAS
  delete Phi_temp;
  hipFree(d_zero) >> GPLDA_CHECK;
  hipFree(d_one) >> GPLDA_CHECK;
  hipblasDestroy(*cublas_handle) >> GPLDA_CHECK;
  delete cublas_handle;

  // remove the args pointer
  args = NULL;
}

extern "C" void sample_phi() {
  // draw Phi ~ PPU(n + beta)
  polya_urn_sample<<<args->K,GPLDA_POLYA_URN_SAMPLE_BLOCKDIM,0,*Phi_stream>>>(Phi->dense, n->dense, args->beta, args->V, pois->pois_alias->prob, pois->pois_alias->alias, pois->max_lambda, pois->max_value, Phi_rng);
  rng_advance<<<1,1,0,*Phi_stream>>>(args->K*args->V,Phi_rng);

  // copy Phi for transpose, set the stream, then transpose Phi
  polya_urn_transpose(Phi_stream, Phi->dense, Phi_temp->dense, args->K, args->V, cublas_handle, d_zero, d_one);

  // compute sigma_a and alias probabilities
  polya_urn_colsums<<<args->V,GPLDA_POLYA_URN_COLSUMS_BLOCKDIM,0,*Phi_stream>>>(Phi->dense, sigma_a, args->alpha, alias->prob, args->K);

  // build Alias tables
  build_alias<<<args->V,32,2*next_pow2(args->K)*sizeof(i32), *Phi_stream>>>(alias->prob, alias->alias, args->K);

  // reset sufficient statistics for n
  polya_urn_reset<<<args->K,128,0,*Phi_stream>>>(n->dense, args->V);

  // don't return until operations completed
  hipStreamSynchronize(*Phi_stream) >> GPLDA_CHECK;
}

extern "C" void sample_z_async(Buffer* buffer) {
  // copy z,w,d to GPU and compute d_idx based on document length
  hipMemcpyAsync(buffer->gpu_z, buffer->z, args->buffer_size, hipMemcpyHostToDevice,*buffer->stream) >> GPLDA_CHECK; // copy z to GPU
  hipMemcpyAsync(buffer->gpu_w, buffer->w, args->buffer_size, hipMemcpyHostToDevice,*buffer->stream) >> GPLDA_CHECK; // copy w to GPU
  hipMemcpyAsync(buffer->gpu_d_len, buffer->d, buffer->n_docs, hipMemcpyHostToDevice,*buffer->stream) >> GPLDA_CHECK;
  hipMemcpyAsync(buffer->gpu_K_d, buffer->K_d, buffer->n_docs, hipMemcpyHostToDevice,*buffer->stream) >> GPLDA_CHECK;
  compute_d_idx<<<1,GPLDA_COMPUTE_D_IDX_BLOCKDIM,0,*buffer->stream>>>(buffer->gpu_d_len, buffer->gpu_d_idx, buffer->n_docs);

  // sample the topic indicators
  warp_sample_topics<<<1,32,0,*buffer->stream>>>(args->buffer_size, buffer->n_docs, buffer->gpu_z, buffer->gpu_w, buffer->gpu_d_len, buffer->gpu_d_idx, buffer->gpu_K_d, buffer->gpu_temp, args->K, args->V, args->max_K_d, Phi->dense, alias->prob, alias->alias, buffer->gpu_rng);
  rng_advance<<<1,1,0,*buffer->stream>>>(2*args->buffer_size,Phi_rng);

  // copy z back to host
  hipMemcpyAsync(buffer->z, buffer->gpu_z, args->buffer_size, hipMemcpyDeviceToHost,*buffer->stream) >> GPLDA_CHECK;
}

extern "C" void sync_buffer(Buffer *buffer) {
  // return when stream has finished
  hipStreamSynchronize(*buffer->stream) >> GPLDA_CHECK;
}

}
