#include "hip/hip_runtime.h"
//#pragma once

#include "types.cuh"
#include "tuning.cuh"
#include <hiprand/hiprand_kernel.h> // need to add -lcurand to nvcc flags

#include <cstdio>
#include "assert.h"

#define GPLDA_HASH_EMPTY 0xfffff // 20 bits
#define GPLDA_HASH_LINE_SIZE 16
#define GPLDA_HASH_MAX_NUM_LINES 6

namespace gplda {

union HashMapEntry {
  #pragma pack(1)
  struct {
    u32 relocate: 1;
    u32 backpointer_hash: 3;
    u32 backpointer_idx: 4;
    u32 key: 20;
    u64 value: 36;
  };
  u64 int_repr;
  HashMapEntry(u64 ir) {
    this->int_repr = ir;
  }
  HashMapEntry(u32 r, u32 bh, u32 bs, u32 k, u64 v) {
    this->relocate = r;
    this->backpointer_hash = bh;
    this->backpointer_idx = bs;
    this->key = k;
    this->value = v;
  }
};

static_assert(sizeof(HashMapEntry) == sizeof(u64), "#pragma pack(1) failed in HashMapEntry");

template<SynchronizationType sync_type>
struct HashMap {
  u32 size;
  u32 max_size;
  HashMapEntry* data;
  HashMapEntry* temp_data;
  HashMapEntry* buffer;
  u32 a;
  u32 b;
  u32 c[GPLDA_HASH_MAX_NUM_LINES - 1];
  u32 needs_rebuild;
  hiprandStatePhilox4_32_10_t* rng;

  __device__ __forceinline__ i32 hash_fn(u32 key) {
    return (a * key + b) % 334214459;
  }

  __device__ __forceinline__ i32 hash_slot(u32 key) {
    return (hash_fn(key) % (size / GPLDA_HASH_LINE_SIZE)) * GPLDA_HASH_LINE_SIZE;
  }

  __device__ __forceinline__ i32 cipher_hash_fn(u32 key) {
    return (c[0] * key + c[1]) % 334214459;
  }

  __device__ __forceinline__ i32 cipher_fn(i32 slot, i32 fn_idx) {
    return fn_idx == 0 ? slot : slot ^ cipher_hash_fn(fn_idx - 1);
  }

  __device__ __forceinline__ i32 cipher_fn_idx(i32 key, u32 slot) {
    #pragma unroll
    for(i32 i = 0; i < GPLDA_HASH_MAX_NUM_LINES; ++i) {
      if(cipher_fn(i, key) == slot) {
        return i;
      }
    }
  }



  __device__ __forceinline__ void sync() {
    if(sync_type == block) {
      __syncthreads();
    }
  }

  __device__ inline void provide_buffer(u64* in_buffer) {
    if(threadIdx.x == 0) {
      buffer = (HashMapEntry*) in_buffer;
    }
    sync();
  }




  __device__ inline void init(void* in_data, u32 in_size, u32 in_max_size, hiprandStatePhilox4_32_10_t* in_rng) {
    // calculate initialization variables common for all threads
    i32 dim = (sync_type == block) ? blockDim.x : warpSize;
    i32 thread_idx = threadIdx.x % dim;

    // set map parameters and calculate random hash functions
    if(thread_idx == 0) {
      // round down to ensure cache alignment
      max_size = (in_max_size / GPLDA_HASH_LINE_SIZE) * GPLDA_HASH_LINE_SIZE;
      size = min((in_size / GPLDA_HASH_LINE_SIZE + 1) * GPLDA_HASH_LINE_SIZE, in_max_size);

      // perform pointer arithmetic
      data = (HashMapEntry*) in_data;
      temp_data = data + max_size; // no sizeof for typed pointer arithmetic
      buffer = temp_data + max_size; // no sizeof for typed pointer arithmetic

      needs_rebuild = 0;
      rng = in_rng; // make sure this->rng is set before use
      a = __float2uint_rz(size * hiprand_uniform(rng));
      b = __float2uint_rz(size * hiprand_uniform(rng));
      #pragma unroll
      for(i32 i = 1; i < GPLDA_HASH_MAX_NUM_LINES; ++i) {
        c[i-1] = __float2uint_rz(size * hiprand_uniform(rng));
      }
    }

    // synchronize to ensure shared memory writes are visible
    sync();

    // set map to empty
    for(i32 offset = 0; offset < size / dim + 1; ++offset) {
      i32 i = offset * dim + thread_idx;
      if(i < size) {
        data[i] = HashMapEntry(0,0,0,GPLDA_HASH_EMPTY,0);
      }
    }

    // set buffer to empty
    for(i32 offset = 0; offset < GPLDA_HASH_LINE_SIZE / dim + 1; ++offset) {
      i32 i = offset * dim + thread_idx;
      if(i < GPLDA_HASH_LINE_SIZE) {
        buffer[i] = HashMapEntry(0,0,0,GPLDA_HASH_EMPTY,0);
      }
    }

    // synchronize to ensure initialization is complete
    sync();
  }





  __device__ inline void rebuild() {

  }





  __device__ inline u32 get2(u32 key) {
    // shuffle key to entire half-warp
    key = __shfl(key, 0, warpSize/2);
    i32 half_lane_idx = threadIdx.x % (warpSize / 2);
    u32 half_lane_mask = 0x0000ffff << (((threadIdx.x % warpSize) / 16) * 4); // 4 if lane >= 16, 0 otherwise

    // check table
    i32 initial_slot = hash_slot(key);
    #pragma unroll
    for(i32 i = 0; i < GPLDA_HASH_MAX_NUM_LINES; ++i) {
      // compute slot and retrieve entry
      i32 slot = cipher_fn(initial_slot, i);
      HashMapEntry entry = data[slot + half_lane_idx];

      // check if we found the key
      u32 found = __ballot(entry.key == key) & half_lane_mask;
      if(found != 0) {
        return __shfl(entry.value, __ffs(found), warpSize/2);
      }

      // check if Robin Hood guarantee indicates no key is present
      u32 no_key = __ballot(entry.key == GPLDA_HASH_EMPTY || cipher_fn_idx(entry.key, slot) > i) & half_lane_mask;
      if(no_key != 0) {
        return 0;
      }
    }

    // ran out of possible slots: key not present
    return 0;
  }

  __device__ inline void try_accumulate2(u32 key, i32 diff) {
    // determine half warp indices
    i32 half_lane_idx = threadIdx.x % (warpSize / 2);
    i32 half_warp_idx = threadIdx.x / (warpSize / 2);
    u32 half_lane_mask = 0x0000ffff << (((threadIdx.x % warpSize) / 16) * 4); // 4 if lane >= 16, 0 otherwise

    // acquire ring buffer location
    i32 ring_buffer_start = 0;

    // build entry to be inserted and shuffle to entire half warp
    HashMapEntry halfwarp_entry = HashMapEntry(0,1,0,key,diff);
    halfwarp_entry.int_repr = __shfl(halfwarp_entry.int_repr, 0, warpSize/2);

    // insert key into buffer
    if(half_lane_idx == 0) {
      i32 buffer_idx = (ring_buffer_start + half_warp_idx) % GPLDA_HASH_LINE_SIZE;
      buffer[buffer_idx] = halfwarp_entry;
      halfwarp_entry.backpointer_hash = 1; // buffer
      halfwarp_entry.backpointer_idx = buffer_idx;
    }

    // forward pass: find empty value, accumulate key if present
    i32 initial_slot = hash_slot(key);
    i32 slot_idx = 0;
    i32 done = false;
    for(i32 i = 0; i < 7 * (32 - __clz(size)); ++i) { // fast log base 2
      // compute slot and retrieve entry
      i32 slot = cipher_fn(initial_slot, slot_idx);
      HashMapEntry thread_entry = data[slot + half_lane_idx];

      // assuming Robin Hood guarantees have not kicked in yet, check if we found the key
      if(halfwarp_entry.backpointer_hash == 1) {
        if(thread_entry.key == key) {
          // key found: set relocate intention, accumulate, clear buffer, and exit if successful
          buffer[halfwarp_entry.backpointer_idx].relocate = true;
          HashMapEntry replacement = thread_entry;
          replacement.value += diff;
          // perform CAS, retrying if necessary
          while(true) {
            HashMapEntry old = HashMapEntry(atomicCAS(&data[slot + half_lane_idx].int_repr, thread_entry.int_repr, replacement.int_repr));
            if(old.int_repr == thread_entry.int_repr) {
              // update was successful: clear buffer
              buffer[halfwarp_entry.backpointer_idx] = HashMapEntry(0,0,0,GPLDA_HASH_EMPTY,0);
              done = true;
              break;
            } else if(old.key != thread_entry.key) {
              // key and value changed: remove relocate intention
              buffer[halfwarp_entry.backpointer_idx].relocate = false;
              break;
            }
            // else value changed but key didn't: try another CAS
          }
        }
        if((__ballot(done == true) & half_lane_mask) != 0) {
          return;
        }
      }

      // key is not present: see if we can take some other key's slot
      i32 thread_entry_initial_slot;
      if(thread_entry.backpointer_hash == 0) {
        // no backpointer: reverse current entry's hash function
        thread_entry_initial_slot = cipher_fn_idx(thread_entry.key, slot);
      } else {
        // follow backpointer, then reverse the hash function
        HashMapEntry thread_backpointer_entry = thread_entry.backpointer_hash == 1 ?
            buffer[thread_entry.backpointer_idx] : // backpointer points to buffer
            data[cipher_fn(slot, thread_entry.backpointer_hash) + thread_entry.backpointer_idx]; // backpointer points to table
        thread_entry_initial_slot = cipher_fn_idx(thread_backpointer_entry.key, slot);
      }

      // increment slot index
      slot_idx++;
      if(slot_idx >= GPLDA_HASH_MAX_NUM_LINES) {
        // no available slot in 6 cache lines: resize table
      }
    }

    // backward pass to insert value
    while(true) {
      break;
    }


  }

  __device__ __forceinline__ void accumulate2(u32 key, i32 diff) {
    // try to accumulate
    try_accumulate2(key, diff);

    // rebuild if too large
    sync();
    if(needs_rebuild == 1) {
      rebuild();
    }
  }
};

}
