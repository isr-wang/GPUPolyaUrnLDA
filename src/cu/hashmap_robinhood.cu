#include "hip/hip_runtime.h"
//#pragma once

#include "types.cuh"
#include "tuning.cuh"
#include <hiprand/hiprand_kernel.h> // need to add -lcurand to nvcc flags

#include <cstdio>
#include "assert.h"

#define GPLDA_HASH_EMPTY 0xfffff // 20 bits
#define GPLDA_HASH_LINE_SIZE 16
#define GPLDA_HASH_MAX_NUM_LINES 6

namespace gplda {

union HashMapEntry {
  struct {
    u32 relocate: 1;
    u32 backpointer_hash: 3;
    u32 backpointer_slot: 4;
    u32 key: 20;
    u64 value: 36;
  };
  u64 int_repr;
  HashMapEntry(u32 r, u32 bh, u32 bs, u32 k, u64 v) {
    this->relocate = r;
    this->backpointer_hash = bh;
    this->backpointer_slot = bs;
    this->key = k;
    this->value = v;
  }
};

template<SynchronizationType sync_type>
struct HashMap {
  u32 size;
  u32 max_size;
  HashMapEntry* data;
  HashMapEntry* temp_data;
  HashMapEntry* buffer;
  u32 a;
  u32 b;
  u32 c[GPLDA_HASH_MAX_NUM_LINES - 1];
  u32 needs_rebuild;
  hiprandStatePhilox4_32_10_t* rng;

  __device__ __forceinline__ u32 left_32_bits(u64 x) {
    return (u32) (x >> 32);
  }

  __device__ __forceinline__ u32 right_32_bits(u64 x) {
    return (u32) x;
  }

  __device__ __forceinline__ i32 hash_fn(u32 key) {
    return (a * key + b) % 334214459;
  }

  __device__ __forceinline__ i32 hash_slot(u32 key) {
    return (hash_fn(key) % (size / GPLDA_HASH_LINE_SIZE)) * GPLDA_HASH_LINE_SIZE;
  }

  __device__ __forceinline__ i32 rev_hash_fn(u32 key, i32 i) {
    return i == 0 ? hash_fn(key) : key ^ c[(((c[0] * key + c[1]) % 334214459) + i - 1) % (GPLDA_HASH_MAX_NUM_LINES - 1)];
  }

  __device__ __forceinline__ i32 rev_hash_fn_idx(u32 key, u32 slot) {
    #pragma unroll
    for(i32 i = 0; i < GPLDA_HASH_MAX_NUM_LINES; ++i) {
      if(rev_hash_fn(key, i) == slot) {
        return i;
      }
    }
  }



  __device__ __forceinline__ void sync() {
    if(sync_type == block) {
      __syncthreads();
    }
  }

  __device__ inline void provide_buffer(u64* in_buffer) {
    if(threadIdx.x == 0) {
      buffer = (HashMapEntry*) in_buffer;
    }
    sync();
  }




  __device__ inline void init(void* in_data, u32 in_size, u32 in_max_size, hiprandStatePhilox4_32_10_t* in_rng) {
    // calculate initialization variables common for all threads
    i32 dim = (sync_type == block) ? blockDim.x : warpSize;
    i32 thread_idx = threadIdx.x % dim;

    // set map parameters and calculate random hash functions
    if(thread_idx == 0) {
      // round down to ensure cache alignment
      max_size = (in_max_size / GPLDA_HASH_LINE_SIZE) * GPLDA_HASH_LINE_SIZE;
      size = min((in_size / GPLDA_HASH_LINE_SIZE + 1) * GPLDA_HASH_LINE_SIZE, in_max_size);

      // perform pointer arithmetic
      data = (HashMapEntry*) in_data;
      temp_data = data + max_size; // no sizeof for typed pointer arithmetic
      buffer = temp_data + max_size; // no sizeof for typed pointer arithmetic

      needs_rebuild = 0;
      rng = in_rng; // make sure this->rng is set before use
      a = __float2uint_rz(size * hiprand_uniform(rng));
      b = __float2uint_rz(size * hiprand_uniform(rng));
      #pragma unroll
      for(i32 i = 1; i < GPLDA_HASH_MAX_NUM_LINES; ++i) {
        c[i-1] = __float2uint_rz(size * hiprand_uniform(rng));
      }
    }

    // synchronize to ensure shared memory writes are visible
    sync();

    // set map to empty
    for(i32 offset = 0; offset < size / dim + 1; ++offset) {
      i32 i = offset * dim + thread_idx;
      if(i < size) {
        data[i] = HashMapEntry(0,0,0,GPLDA_HASH_EMPTY,0);
      }
    }

    // set buffer to empty
    for(i32 offset = 0; offset < GPLDA_HASH_LINE_SIZE / dim + 1; ++offset) {
      i32 i = offset * dim + thread_idx;
      if(i < GPLDA_HASH_LINE_SIZE) {
        buffer[i] = HashMapEntry(0,0,0,GPLDA_HASH_EMPTY,0);
      }
    }

    // synchronize to ensure initialization is complete
    sync();
  }





  __device__ inline void rebuild() {

  }





  __device__ inline u32 get2(u32 key) {
    // shuffle key to entire half-warp
    key = __shfl(key, 0, warpSize/2);
    i32 half_lane_idx = threadIdx.x % (warpSize / 2);
    u32 half_lane_mask = 0x0000ffff << (((threadIdx.x % warpSize) / 16) * 4); // 4 if lane >= 16, 0 otherwise

    // check table
    i32 initial_slot = hash_slot(key);
    #pragma unroll
    for(i32 i = 0; i < GPLDA_HASH_MAX_NUM_LINES; ++i) {
      // compute slot
      i32 slot = rev_hash_fn(initial_slot, i);

      HashMapEntry entry = data[slot + half_lane_idx];

      // check if we found the key
      u32 found = __ballot(entry.key == key) & half_lane_mask;
      if(found != 0) {
        return __shfl(entry.value, __ffs(found), warpSize/2);
      }

      // check if Robin Hood guarantee indicates no key is present
      u32 no_key = __ballot(entry.key == GPLDA_HASH_EMPTY || rev_hash_fn_idx(entry.key, slot) > i) & half_lane_mask;
      if(no_key != 0) {
        return 0;
      }
    }

    // ran out of possible slots: key not present
    return 0;
  }

  __device__ inline void try_accumulate2(u32 key, u32 diff) {
    // shuffle key and diff to entire half warp
    key = __shfl(key, 0, warpSize/2);
    diff = __shfl(diff, 0, warpSize/2);
    i32 half_lane_idx = threadIdx.x % (warpSize / 2);
    i32 half_warp_idx = threadIdx.x / (warpSize / 2);
    u32 half_lane_mask = 0x0000ffff << (((threadIdx.x % warpSize) / 16) * 4); // 4 if lane >= 16, 0 otherwise
    i32 ring_buffer_start = 0;
    i32 backpointer_hash;
    i32 backpointer_slot;

    // insert key into buffer
    if(half_lane_idx == 0) {
      backpointer_hash = 1; // buffer
      backpointer_slot = (ring_buffer_start + half_warp_idx) % GPLDA_HASH_MAX_NUM_LINES;
      buffer[backpointer_slot] = HashMapEntry(0,0,0,key,diff);
    }

    // forward pass to find empty value, accumulate key if present

    // backward pass to insert value

  }

  __device__ __forceinline__ void accumulate2(u32 key, u32 diff) {
    // try to accumulate
    try_accumulate2(key, diff);

    // rebuild if too large
    sync();
    if(needs_rebuild == 1) {
      rebuild();
    }
  }
};

}
