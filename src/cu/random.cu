#include "hip/hip_runtime.h"
#include "random.cuh"

namespace gplda {

// initializer for random number generator
__global__ void rng_init(u32 seed, u32 subsequence, hiprandStatePhilox4_32_10_t* rng) {
  if(threadIdx.x == 0 && blockIdx.x == 0) {
    hiprand_init((unsigned long long) seed, (unsigned long long) subsequence, (unsigned long long) 0, rng);
  }
}

// advance for random number generator
__global__ void rng_advance(u32 advance, hiprandStatePhilox4_32_10_t* rng) {
  if(threadIdx.x == 0 && blockIdx.x == 0) {
    skipahead((unsigned long long) advance, rng);
  }
}

}
