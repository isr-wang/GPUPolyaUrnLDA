#include "hip/hip_runtime.h"
#include "error.h"
#include "poisson.h"
#include "train.h"

namespace gplda {

__global__ void build_poisson(float** prob, float** alias, float beta, size_t lambda, size_t size) {
}

__global__ void draw_poisson(float** prob, float** alias, size_t* lambda, size_t n) {
}

Poisson::Poisson(size_t ml, size_t mv) {
  // assign class parameters
  max_lambda = ml;
  max_value = mv;
  // allocate array of pointers on host first, so hipMalloc can populate it
  float** prob_host = new float*[max_lambda];
  float** alias_host = new float*[max_lambda];
  // allocate each Alias table
  for(size_t i = 0; i < max_lambda; ++i) {
    hipMalloc(&prob_host[i], max_value * sizeof(float)) >> GPLDA_CHECK;
    hipMalloc(&alias_host[i], max_value * sizeof(float)) >> GPLDA_CHECK;
  }
  // now, allocate array of pointers on device
  hipMalloc(&prob, max_lambda * sizeof(float*)) >> GPLDA_CHECK;
  hipMalloc(&alias, max_lambda * sizeof(float*)) >> GPLDA_CHECK;
  // copy array of pointers to device
  hipMemcpy(prob, prob_host, max_lambda * sizeof(float*), hipMemcpyHostToDevice) >> GPLDA_CHECK;
  hipMemcpy(alias, alias_host, max_lambda * sizeof(float*), hipMemcpyHostToDevice) >> GPLDA_CHECK;
  // deallocate array of pointers on host
  delete[] prob_host;
  delete[] alias_host;
  // launch kernel to build the alias tables
  build_poisson<<<max_lambda,1>>>(prob, alias, ARGS->beta, max_lambda, max_value);
}

Poisson::~Poisson() {
  // allocate array of pointers on host, so we can dereference it
  float** prob_host = new float*[max_lambda];
  float** alias_host = new float*[max_lambda];
  // copy array of pointers to host
  hipMemcpy(prob_host, prob, max_lambda * sizeof(float*), hipMemcpyDeviceToHost) >> GPLDA_CHECK;
  hipMemcpy(alias_host, alias, max_lambda * sizeof(float*), hipMemcpyDeviceToHost) >> GPLDA_CHECK;
  // free the memory at the arrays being pointed to
  for(size_t i = 0; i < max_lambda; ++i) {
    hipFree(prob_host[i]) >> GPLDA_CHECK;
    hipFree(alias_host[i]) >> GPLDA_CHECK;
  }
  // free the memory of the pointer array on device
  hipFree(prob) >> GPLDA_CHECK;
  hipFree(alias) >> GPLDA_CHECK;
  // deallocate array of pointers on host
  delete[] prob_host;
  delete[] alias_host;
}

}
