#include "hip/hip_runtime.h"
#include "assert.h"
#include "error.cuh"
#include "poisson.cuh"
#include "train.cuh"

namespace gplda {

__global__ void build_poisson(float** prob, float** alias, float beta, int table_size) {
  assert(blockDim.x == 32); // for simplicity, Poisson Alias tables are built on the warp level, so exit if misconfigured
  int lambda = blockIdx.x; // each block builds one table
  float L = lambda + beta;
  // populate PMF
  for(int offset = 0; offset < table_size / blockDim.x + 1; ++offset) {
    int i = threadIdx.x + offset * blockDim.x;
    float x = i;
    if(i < table_size) {
      prob[lambda][i] = expf(x*logf(L) - L - lgammaf(x + 1));
    }
  }
  __syncthreads();
  // build array of large probabilities
  extern __shared__ float large[];
  __shared__ int num_large[1];
  float cutoff = 1.0/((float) table_size);
  // loop over PMF
  for(int offset = 0; offset < table_size / blockDim.x + 1; ++offset) {
    int i = threadIdx.x + offset * blockDim.x;
    // determine which warps have large probabilities
    unsigned int warp_large = __ballot(prob[lambda][i] > cutoff);
    // determine how many large probabilities are in the warp's view
    int warp_num_large = __popc(warp_large);
    // increment the array's size
    int large_start = atomicAdd(num_large, warp_num_large);
    // if current warp has elements, add elements to the array
    if(1/*warp_bit_set*/) {
      large[large_start + 0 /*warp_bit_offset*/] = prob[lambda][i];
    }
  }
  // we've now built large array, let's grab elements and place them

}

__global__ void draw_poisson(float** prob, float** alias, int* lambda, int n) {
}

Poisson::Poisson(int ml, int mv) {
  // assign class parameters
  max_lambda = ml;
  max_value = mv;
  // allocate array of pointers on host first, so hipMalloc can populate it
  float** prob_host = new float*[max_lambda];
  float** alias_host = new float*[max_lambda];
  // allocate each Alias table
  for(size_t i = 0; i < max_lambda; ++i) {
    hipMalloc(&prob_host[i], max_value * sizeof(float)) >> GPLDA_CHECK;
    hipMalloc(&alias_host[i], max_value * sizeof(float)) >> GPLDA_CHECK;
  }
  // now, allocate array of pointers on device
  hipMalloc(&prob, max_lambda * sizeof(float*)) >> GPLDA_CHECK;
  hipMalloc(&alias, max_lambda * sizeof(float*)) >> GPLDA_CHECK;
  // copy array of pointers to device
  hipMemcpy(prob, prob_host, max_lambda * sizeof(float*), hipMemcpyHostToDevice) >> GPLDA_CHECK;
  hipMemcpy(alias, alias_host, max_lambda * sizeof(float*), hipMemcpyHostToDevice) >> GPLDA_CHECK;
  // deallocate array of pointers on host
  delete[] prob_host;
  delete[] alias_host;
  // launch kernel to build the alias tables
  build_poisson<<<max_lambda,32>>>(prob, alias, ARGS->beta, max_value);
}

Poisson::~Poisson() {
  // allocate array of pointers on host, so we can dereference it
  float** prob_host = new float*[max_lambda];
  float** alias_host = new float*[max_lambda];
  // copy array of pointers to host
  hipMemcpy(prob_host, prob, max_lambda * sizeof(float*), hipMemcpyDeviceToHost) >> GPLDA_CHECK;
  hipMemcpy(alias_host, alias, max_lambda * sizeof(float*), hipMemcpyDeviceToHost) >> GPLDA_CHECK;
  // free the memory at the arrays being pointed to
  for(size_t i = 0; i < max_lambda; ++i) {
    hipFree(prob_host[i]) >> GPLDA_CHECK;
    hipFree(alias_host[i]) >> GPLDA_CHECK;
  }
  // free the memory of the pointer array on device
  hipFree(prob) >> GPLDA_CHECK;
  hipFree(alias) >> GPLDA_CHECK;
  // deallocate array of pointers on host
  delete[] prob_host;
  delete[] alias_host;
}

}
