#include "hip/hip_runtime.h"
#include "assert.h"
#include "error.cuh"
#include "poisson.cuh"
#include "train.cuh"

namespace gplda {


__device__ __forceinline__ unsigned int lane_id_bits(int thread_idx) {
  return ((unsigned int) 1) << (thread_idx % warpSize);
}

__device__ __forceinline__ unsigned int lane_offset(unsigned int lane_bits, int thread_idx) {
  return __popc((~(((unsigned int) 4294967295) << (thread_idx % warpSize))) & lane_bits);
}

__global__ void build_poisson(float** prob, float** alias, float beta, int table_size) {
//  assert(blockDim.x == 32); // for simplicity, Poisson Alias tables are built on the warp level, so exit if misconfigured
  int lambda = blockIdx.x; // each block builds one table
  float L = lambda + beta;
  // populate PMF
  for(int offset = 0; offset < table_size / blockDim.x + 1; ++offset) {
    int i = threadIdx.x + offset * blockDim.x;
    float x = i;
    if(i < table_size) {
      prob[lambda][i] = expf(x*logf(L) - L - lgammaf(x + 1));
    }
  }
  __syncthreads();
  // build array of large probabilities
  /*extern*/ __shared__ int large[200];
  __shared__ int num_large[1];
  if(threadIdx.x == 0) {
    num_large[0] = 0;
  }
  __syncthreads();
  float cutoff = 1.0/((float) table_size);
  // loop over PMF
  for(int offset = 0; offset < table_size / blockDim.x + 1; ++offset) {
    int i = threadIdx.x + offset * blockDim.x;
    if(i < table_size) {
      float thread_prob = prob[lambda][i];
      // determine which threads have large probabilities
      unsigned int warp_large_bits = __ballot(thread_prob >= cutoff);
      // determine how many large probabilities are in the warp's view
      int warp_num_large = __popc(warp_large_bits);
      // increment the array's size, only once per warp, then broadcast to all lanes in the warp
      int warp_large_start;
      if(threadIdx.x % warpSize == 0) {
        warp_large_start = atomicAdd(num_large, warp_num_large);
      }
      warp_large_start = __shfl(warp_large_start, 0);
      // if current warp has elements, add elements to the array
      if(thread_prob >= cutoff) {
        large[warp_large_start + lane_offset(warp_large_bits, threadIdx.x)] = i;
      }
    }
  }
  __syncthreads();
  // grab a set of indices from large array for the warp to work on

  // loop over each warp's range

    // try to place probabilities into current window

    // place any small probabilities into slots and small index array, grab new probability

    // if large stack empty, perform a warp rebalance

    // if large stack empty and cannot rebalance, write current index to window stack

  __syncthreads();
  // if still holding indices, grab an index from window stack and iterate over that


  __syncthreads();
  // if window stack empty, grab a set of elements from small stack and iterate over that

  __syncthreads();
  // at this point, all remaining slots must have probability 1
}


__global__ void draw_poisson(float** prob, float** alias, int* lambda, int n) {
}

Poisson::Poisson(int ml, int mv) {
  // assign class parameters
  max_lambda = ml;
  max_value = mv;
  // allocate array of pointers on host first, so hipMalloc can populate it
  float** prob_host = new float*[max_lambda];
  float** alias_host = new float*[max_lambda];
  // allocate each Alias table
  for(size_t i = 0; i < max_lambda; ++i) {
    hipMalloc(&prob_host[i], max_value * sizeof(float)) >> GPLDA_CHECK;
    hipMalloc(&alias_host[i], max_value * sizeof(float)) >> GPLDA_CHECK;
  }
  // now, allocate array of pointers on device
  hipMalloc(&prob, max_lambda * sizeof(float*)) >> GPLDA_CHECK;
  hipMalloc(&alias, max_lambda * sizeof(float*)) >> GPLDA_CHECK;
  // copy array of pointers to device
  hipMemcpy(prob, prob_host, max_lambda * sizeof(float*), hipMemcpyHostToDevice) >> GPLDA_CHECK;
  hipMemcpy(alias, alias_host, max_lambda * sizeof(float*), hipMemcpyHostToDevice) >> GPLDA_CHECK;
  // deallocate array of pointers on host
  delete[] prob_host;
  delete[] alias_host;
  // launch kernel to build the alias tables
  build_poisson<<<max_lambda,64/*32*/,max_value*sizeof(int)>>>(prob, alias, ARGS->beta, max_value);
  hipDeviceSynchronize();
}

Poisson::~Poisson() {
  // allocate array of pointers on host, so we can dereference it
  float** prob_host = new float*[max_lambda];
  float** alias_host = new float*[max_lambda];
  // copy array of pointers to host
  hipMemcpy(prob_host, prob, max_lambda * sizeof(float*), hipMemcpyDeviceToHost) >> GPLDA_CHECK;
  hipMemcpy(alias_host, alias, max_lambda * sizeof(float*), hipMemcpyDeviceToHost) >> GPLDA_CHECK;
  // free the memory at the arrays being pointed to
  for(size_t i = 0; i < max_lambda; ++i) {
    hipFree(prob_host[i]) >> GPLDA_CHECK;
    hipFree(alias_host[i]) >> GPLDA_CHECK;
  }
  // free the memory of the pointer array on device
  hipFree(prob) >> GPLDA_CHECK;
  hipFree(alias) >> GPLDA_CHECK;
  // deallocate array of pointers on host
  delete[] prob_host;
  delete[] alias_host;
}

}
