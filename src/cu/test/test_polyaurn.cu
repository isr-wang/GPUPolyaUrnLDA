#include "test_polyaurn.cuh"
#include "../polyaurn.cuh"
#include "../error.cuh"
#include "assert.h"

using gplda::FileLine;

namespace gplda_test {

void test_polya_urn_init() {

}

void test_polya_urn_sample() {

}

void test_polya_urn_transpose() {
  // 0.3 0.3 0.4
  // 0.2 0.5 0.3
  // 0.1 0.1 0.8
  float Phi_host[9] = {0.3f, 0.3f, 0.4f, 0.2f, 0.5f, 0.3f, 0.1f, 0.1f, 0.8f};
  float* Phi;
  float* Phi_temp;

  hipMalloc(&Phi, 9 * sizeof(float)) >> GPLDA_CHECK;
  hipMalloc(&Phi_temp, 9 * sizeof(float)) >> GPLDA_CHECK;

  hipMemcpy(Phi, Phi_host, 9 * sizeof(float), hipMemcpyHostToDevice) >> GPLDA_CHECK;

  hipStream_t* stream = new hipStream_t;
  hipStreamCreate(stream) >> GPLDA_CHECK;

  hipblasHandle_t* cublas_handle = new hipblasHandle_t;
  hipblasCreate(cublas_handle) >> GPLDA_CHECK;
  hipblasSetPointerMode(*cublas_handle, HIPBLAS_POINTER_MODE_DEVICE) >> GPLDA_CHECK;

  float* d_zero;
  float* d_one;
  hipMalloc(&d_zero, sizeof(float)) >> GPLDA_CHECK;
  hipMemset(d_zero, 0.0f, sizeof(float)) >> GPLDA_CHECK;
  hipMalloc(&d_one, sizeof(float)) >> GPLDA_CHECK;
  hipMemset(d_one, 1.0f, sizeof(float)) >> GPLDA_CHECK;

  gplda::polya_urn_transpose(stream, Phi, Phi_temp, 3, 3, cublas_handle, d_zero, d_one);
  hipStreamSynchronize(*stream);

  hipMemcpy(Phi_host, Phi, 9 * sizeof(float), hipMemcpyDeviceToHost);

  assert(Phi_host[0] == 0.3f);
  assert(Phi_host[1] == 0.2f);
  assert(Phi_host[2] == 0.1f);
  assert(Phi_host[3] == 0.3f);
  assert(Phi_host[4] == 0.5f);
  assert(Phi_host[5] == 0.1f);
  assert(Phi_host[6] == 0.4f);
  assert(Phi_host[7] == 0.3f);
  assert(Phi_host[8] == 0.8f);

  hipStreamDestroy(*stream);
  delete stream;

  hipblasDestroy(*cublas_handle);
  delete cublas_handle;

  hipFree(d_zero);
  hipFree(d_one);
  hipFree(Phi);
  hipFree(Phi_temp);
}

void test_polya_urn_colsums() {
  float tolerance = 0.0001f;
  // 0.3 0.3 0.4
  // 0.2 0.5 0.3
  // 0.1 0.1 0.8
  float Phi_host[9] = {0.3f, 0.2f, 0.1f, 0.3f, 0.5f, 0.1f, 0.4f, 0.3f, 0.8f};
  float* Phi;

  hipMalloc(&Phi, 9 * sizeof(float)) >> GPLDA_CHECK;

  hipMemcpy(Phi, Phi_host, 9 * sizeof(float), hipMemcpyHostToDevice) >> GPLDA_CHECK;

  float* sigma_a;
  hipMalloc(&sigma_a, 3 * sizeof(float)) >> GPLDA_CHECK;

  float** prob;
  hipMalloc(&prob, 3 * sizeof(float*)) >> GPLDA_CHECK;

  float* prob_1;
  float* prob_2;
  float* prob_3;
  hipMalloc(&prob_1, 3 * sizeof(float)) >> GPLDA_CHECK;
  hipMalloc(&prob_2, 3 * sizeof(float)) >> GPLDA_CHECK;
  hipMalloc(&prob_3, 3 * sizeof(float)) >> GPLDA_CHECK;

  float* prob_host[3] = {prob_1, prob_2, prob_3};

  hipMemcpy(prob, prob_host, 3 * sizeof(float*), hipMemcpyHostToDevice) >> GPLDA_CHECK;

  gplda::polya_urn_colsums<<<3,32>>>(Phi, sigma_a, 1.0f, prob, 3);
  hipDeviceSynchronize() >> GPLDA_CHECK;

  float sigma_a_host[3];

  hipMemcpy(sigma_a_host, sigma_a, 3 * sizeof(float), hipMemcpyDeviceToHost) >> GPLDA_CHECK;

  assert(sigma_a_host[0] - (0.3f + 0.2f + 0.1f) < tolerance);
  assert(sigma_a_host[1] - (0.3f + 0.5f + 0.1f) < tolerance);
  assert(sigma_a_host[2] - (0.4f + 0.3f + 0.8f) < tolerance);

  float prob_host_1[3];
  float prob_host_2[3];
  float prob_host_3[3];

  hipMemcpy(prob_host_1, prob_1, 3 * sizeof(float), hipMemcpyDeviceToHost) >> GPLDA_CHECK;
  hipMemcpy(prob_host_2, prob_2, 3 * sizeof(float), hipMemcpyDeviceToHost) >> GPLDA_CHECK;
  hipMemcpy(prob_host_3, prob_3, 3 * sizeof(float), hipMemcpyDeviceToHost) >> GPLDA_CHECK;

  assert(prob_host_1[0] - (0.3f / (0.3f + 0.2f + 0.1f)) < tolerance);
  assert(prob_host_1[1] - (0.2f / (0.3f + 0.2f + 0.1f)) < tolerance);
  assert(prob_host_1[2] - (0.1f / (0.3f + 0.2f + 0.1f)) < tolerance);

  assert(prob_host_2[0] - (0.3f / (0.3f + 0.5f + 0.1f)) < tolerance);
  assert(prob_host_2[1] - (0.5f / (0.3f + 0.5f + 0.1f)) < tolerance);
  assert(prob_host_2[2] - (0.1f / (0.3f + 0.5f + 0.1f)) < tolerance);

  assert(prob_host_3[0] - (0.4f / (0.4f + 0.3f + 0.8f)) < tolerance);
  assert(prob_host_3[1] - (0.3f / (0.4f + 0.3f + 0.8f)) < tolerance);
  assert(prob_host_3[2] - (0.8f / (0.4f + 0.3f + 0.8f)) < tolerance);

  hipFree(Phi) >> GPLDA_CHECK;
  hipFree(sigma_a) >> GPLDA_CHECK;
  hipFree(prob) >> GPLDA_CHECK;
  hipFree(prob_1) >> GPLDA_CHECK;
  hipFree(prob_2) >> GPLDA_CHECK;
  hipFree(prob_3) >> GPLDA_CHECK;
}

}
