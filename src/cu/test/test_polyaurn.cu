#include "hip/hip_runtime.h"
#include "test_polyaurn.cuh"
#include "../poisson.cuh"
#include "../polyaurn.cuh"
#include "../random.cuh"
#include "../error.cuh"
#include "assert.h"

using gplda::FileLine;

namespace gplda_test {

void test_polya_urn_init() {
  uint32_t K = 1000;
  uint32_t V = 5;
  float beta = 0.01;
  uint32_t n_host[5*1000];
  uint32_t C_host[5] = {1*K, 10*K, 100*K, 1000*K, 10000*K}; // K=3 so E(n_k) = [1, 10, 100, 1000, 10000]
  uint32_t n_sum[5] = {0,0,0,0,0};
  uint32_t n_ssq[5] = {0,0,0,0,0};

  uint32_t* n;
  hipMalloc(&n, K * V * sizeof(uint32_t)) >> GPLDA_CHECK;

  uint32_t* C;
  hipMalloc(&C, V * sizeof(uint32_t)) >> GPLDA_CHECK;
  hipMemcpy(C, C_host, V * sizeof(uint32_t), hipMemcpyHostToDevice) >> GPLDA_CHECK;

  hiprandStatePhilox4_32_10_t* Phi_rng;
  hipMalloc(&Phi_rng, sizeof(hiprandStatePhilox4_32_10_t)) >> GPLDA_CHECK;
  gplda::rand_init<<<1,1>>>(0,0,Phi_rng);
  hipDeviceSynchronize() >> GPLDA_CHECK;

  gplda::Poisson* pois = new gplda::Poisson(100, 200, beta);

  gplda::polya_urn_init<<<K,32>>>(n, C, beta, V, pois->pois_alias->prob, pois->pois_alias->alias, pois->max_lambda, pois->max_value, Phi_rng);
  hipDeviceSynchronize() >> GPLDA_CHECK;

  hipMemcpy(n_host, n, K * V * sizeof(uint32_t), hipMemcpyDeviceToHost);

  // check mean by computing sum
  for(int32_t j = 0; j < K; ++j) {
    for(int32_t i = 0; i < V; ++i) {
      n_sum[i] += n_host[j*V + i];
    }
  }

  // check var by computing sum square
  for(int32_t j = 0; j < K; ++j) {
    for(int32_t i = 0; i < V; ++i) {
      n_ssq[i] += ((n_host[j*V + i] - (n_sum[i] / K)) * (n_host[j*V + i] - (n_sum[i] / K)));
    }
  }

  assert(n_sum[0] / K <= 2);
  assert(n_sum[1] / K >= 9 && n_sum[1] / K <= 11);
  assert(n_sum[2] / K >= 90 && n_sum[2] / K <= 110);
  assert(n_sum[3] / K >= 900 && n_sum[3] / K <= 1100);
  assert(n_sum[4] / K >= 9000 && n_sum[4] / K <= 11000);

  assert(n_ssq[0] / K <= 2);
  assert(n_ssq[1] / K >= 9 && n_ssq[1] / K <= 11);
  assert(n_ssq[2] / K >= 90 && n_ssq[2] / K <= 110);
  assert(n_ssq[3] / K >= 900 && n_ssq[3] / K <= 1100);
  assert(n_ssq[4] / K >= 9000 && n_ssq[4] / K <= 11000);

  hipFree(n);
  hipFree(C);
  hipFree(Phi_rng);
  delete pois;
}

void test_polya_urn_sample() {
  float tolerance = 0.02f; // large to allow for randomness

  uint32_t n_host[9] = {1,10,100,1,1,1,1000,1000,1000};
  float Phi_host[9];

  float* Phi;
  hipMalloc(&Phi, 9 * sizeof(float)) >> GPLDA_CHECK;

  uint32_t* n;
  hipMalloc(&n, 9 * sizeof(uint32_t)) >> GPLDA_CHECK;

  hipMemcpy(n, n_host, 9 * sizeof(uint32_t), hipMemcpyHostToDevice) >> GPLDA_CHECK;

  hiprandStatePhilox4_32_10_t* Phi_rng;
  hipMalloc(&Phi_rng, sizeof(hiprandStatePhilox4_32_10_t)) >> GPLDA_CHECK;
  gplda::rand_init<<<1,1>>>(0,0,Phi_rng);
  hipDeviceSynchronize() >> GPLDA_CHECK;

  gplda::Poisson* pois = new gplda::Poisson(100, 200, 0.01f);

  gplda::polya_urn_sample<<<3,32>>>(Phi, n, 0.01f, 3, pois->pois_alias->prob, pois->pois_alias->alias, pois->max_lambda, pois->max_value, Phi_rng);
  hipDeviceSynchronize() >> GPLDA_CHECK;

  hipMemcpy(Phi_host, Phi, 9 * sizeof(float), hipMemcpyDeviceToHost) >> GPLDA_CHECK;

  assert(abs(Phi_host[0] - 0.01f) < tolerance);
  assert(abs(Phi_host[1] - 0.09f) < tolerance);
  assert(abs(Phi_host[2] - 0.9f) < tolerance);
  assert(abs(Phi_host[3] - 0.5f) < tolerance);
  assert(abs(Phi_host[4] - 0.0f) < tolerance);
  assert(abs(Phi_host[5] - 0.5f) < tolerance);
  assert(abs(Phi_host[6] - 0.33f) < tolerance);
  assert(abs(Phi_host[7] - 0.33f) < tolerance);
  assert(abs(Phi_host[8] - 0.33f) < tolerance);

  hipFree(Phi);
  hipFree(n);
  hipFree(Phi_rng);
  delete pois;

}

void test_polya_urn_transpose() {
  // 0.3 0.3 0.4
  // 0.2 0.5 0.3
  // 0.1 0.1 0.8
  float Phi_host[9] = {0.3f, 0.3f, 0.4f, 0.2f, 0.5f, 0.3f, 0.1f, 0.1f, 0.8f};
  float* Phi;
  float* Phi_temp;

  hipMalloc(&Phi, 9 * sizeof(float)) >> GPLDA_CHECK;
  hipMalloc(&Phi_temp, 9 * sizeof(float)) >> GPLDA_CHECK;

  hipMemcpy(Phi, Phi_host, 9 * sizeof(float), hipMemcpyHostToDevice) >> GPLDA_CHECK;

  hipStream_t* stream = new hipStream_t;
  hipStreamCreate(stream) >> GPLDA_CHECK;

  hipblasHandle_t* cublas_handle = new hipblasHandle_t;
  hipblasCreate(cublas_handle) >> GPLDA_CHECK;
  hipblasSetPointerMode(*cublas_handle, HIPBLAS_POINTER_MODE_DEVICE) >> GPLDA_CHECK;

  float h_one = 1.0f; // hipMemset for some reason doesn't work correctly
  float h_zero = 0.0f;
  float* d_zero;
  float* d_one;
  hipMalloc(&d_zero, sizeof(float)) >> GPLDA_CHECK;
  hipMemcpy(d_zero, &h_zero, sizeof(float), hipMemcpyHostToDevice) >> GPLDA_CHECK;
  hipMalloc(&d_one, sizeof(float)) >> GPLDA_CHECK;
  hipMemcpy(d_one, &h_one, sizeof(float), hipMemcpyHostToDevice) >> GPLDA_CHECK;

  gplda::polya_urn_transpose(stream, Phi, Phi_temp, 3, 3, cublas_handle, d_zero, d_one);
  hipStreamSynchronize(*stream);

  hipMemcpy(Phi_host, Phi, 9 * sizeof(float), hipMemcpyDeviceToHost);

  assert(Phi_host[0] == 0.3f);
  assert(Phi_host[1] == 0.2f);
  assert(Phi_host[2] == 0.1f);
  assert(Phi_host[3] == 0.3f);
  assert(Phi_host[4] == 0.5f);
  assert(Phi_host[5] == 0.1f);
  assert(Phi_host[6] == 0.4f);
  assert(Phi_host[7] == 0.3f);
  assert(Phi_host[8] == 0.8f);

  hipStreamDestroy(*stream);
  delete stream;

  hipblasDestroy(*cublas_handle);
  delete cublas_handle;

  hipFree(d_zero);
  hipFree(d_one);
  hipFree(Phi);
  hipFree(Phi_temp);
}

void test_polya_urn_reset() {
  uint32_t n_host[9] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
  uint32_t* n;

  hipMalloc(&n, 9 * sizeof(uint32_t)) >> GPLDA_CHECK;
  hipMemcpy(n, n_host, 9 * sizeof(uint32_t), hipMemcpyHostToDevice) >> GPLDA_CHECK;

  gplda::polya_urn_reset<<<3, 128>>>(n, 3);
  hipDeviceSynchronize() >> GPLDA_CHECK;

  hipMemcpy(n_host, n, 9 * sizeof(uint32_t), hipMemcpyDeviceToHost);

  assert(n_host[0] == 0);
  assert(n_host[1] == 0);
  assert(n_host[2] == 0);
  assert(n_host[3] == 0);
  assert(n_host[4] == 0);
  assert(n_host[5] == 0);
  assert(n_host[6] == 0);
  assert(n_host[7] == 0);
  assert(n_host[8] == 0);

  hipFree(n);
}

void test_polya_urn_colsums() {
  float tolerance = 0.0001f;
  // 0.3 0.3 0.4
  // 0.2 0.5 0.3
  // 0.1 0.1 0.8
  float Phi_host[9] = {0.3f, 0.2f, 0.1f, 0.3f, 0.5f, 0.1f, 0.4f, 0.3f, 0.8f};
  float* Phi;

  hipMalloc(&Phi, 9 * sizeof(float)) >> GPLDA_CHECK;

  hipMemcpy(Phi, Phi_host, 9 * sizeof(float), hipMemcpyHostToDevice) >> GPLDA_CHECK;

  float* sigma_a;
  hipMalloc(&sigma_a, 3 * sizeof(float)) >> GPLDA_CHECK;

  float** prob;
  hipMalloc(&prob, 3 * sizeof(float*)) >> GPLDA_CHECK;

  float* prob_1;
  float* prob_2;
  float* prob_3;
  hipMalloc(&prob_1, 3 * sizeof(float)) >> GPLDA_CHECK;
  hipMalloc(&prob_2, 3 * sizeof(float)) >> GPLDA_CHECK;
  hipMalloc(&prob_3, 3 * sizeof(float)) >> GPLDA_CHECK;

  float* prob_host[3] = {prob_1, prob_2, prob_3};

  hipMemcpy(prob, prob_host, 3 * sizeof(float*), hipMemcpyHostToDevice) >> GPLDA_CHECK;

  gplda::polya_urn_colsums<<<3,32>>>(Phi, sigma_a, 1.0f, prob, 3);
  hipDeviceSynchronize() >> GPLDA_CHECK;

  float sigma_a_host[3];

  hipMemcpy(sigma_a_host, sigma_a, 3 * sizeof(float), hipMemcpyDeviceToHost) >> GPLDA_CHECK;

  assert(sigma_a_host[0] - (0.3f + 0.2f + 0.1f) < tolerance);
  assert(sigma_a_host[1] - (0.3f + 0.5f + 0.1f) < tolerance);
  assert(sigma_a_host[2] - (0.4f + 0.3f + 0.8f) < tolerance);

  float prob_host_1[3];
  float prob_host_2[3];
  float prob_host_3[3];

  hipMemcpy(prob_host_1, prob_1, 3 * sizeof(float), hipMemcpyDeviceToHost) >> GPLDA_CHECK;
  hipMemcpy(prob_host_2, prob_2, 3 * sizeof(float), hipMemcpyDeviceToHost) >> GPLDA_CHECK;
  hipMemcpy(prob_host_3, prob_3, 3 * sizeof(float), hipMemcpyDeviceToHost) >> GPLDA_CHECK;

  assert(abs(prob_host_1[0] - (0.3f / (0.3f + 0.2f + 0.1f))) < tolerance);
  assert(abs(prob_host_1[1] - (0.2f / (0.3f + 0.2f + 0.1f))) < tolerance);
  assert(abs(prob_host_1[2] - (0.1f / (0.3f + 0.2f + 0.1f))) < tolerance);

  assert(abs(prob_host_2[0] - (0.3f / (0.3f + 0.5f + 0.1f))) < tolerance);
  assert(abs(prob_host_2[1] - (0.5f / (0.3f + 0.5f + 0.1f))) < tolerance);
  assert(abs(prob_host_2[2] - (0.1f / (0.3f + 0.5f + 0.1f))) < tolerance);

  assert(abs(prob_host_3[0] - (0.4f / (0.4f + 0.3f + 0.8f))) < tolerance);
  assert(abs(prob_host_3[1] - (0.3f / (0.4f + 0.3f + 0.8f))) < tolerance);
  assert(abs(prob_host_3[2] - (0.8f / (0.4f + 0.3f + 0.8f))) < tolerance);

  hipFree(Phi) >> GPLDA_CHECK;
  hipFree(sigma_a) >> GPLDA_CHECK;
  hipFree(prob) >> GPLDA_CHECK;
  hipFree(prob_1) >> GPLDA_CHECK;
  hipFree(prob_2) >> GPLDA_CHECK;
  hipFree(prob_3) >> GPLDA_CHECK;
}

}
