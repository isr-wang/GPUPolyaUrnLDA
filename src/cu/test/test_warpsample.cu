#include "test_warpsample.cuh"
#include "../warpsample.cuh"
#include "../error.cuh"
#include "assert.h"

using gplda::FileLine;

namespace gplda_test {

void test_compute_d_idx() {
  uint32_t size = 15;
  uint32_t d_len[15] = {3,2,4,2,0, 0,0,0,0,0, 0,0,0,0,0};
  uint32_t d_idx[15] = {0,0,0,0,0, 0,0,0,0,0, 0,0,0,0,0};
  uint32_t n_docs = 4;

  uint32_t* gpu_d_len;
  uint32_t* gpu_d_idx;
  hipMalloc(&gpu_d_len, size*sizeof(uint32_t)) >> GPLDA_CHECK;
  hipMalloc(&gpu_d_idx, size*sizeof(uint32_t)) >> GPLDA_CHECK;

  hipMemcpy(gpu_d_len, d_len, size*sizeof(uint32_t), hipMemcpyHostToDevice) >> GPLDA_CHECK;

  hipStream_t* stream = new hipStream_t;
  hipStreamCreate(stream) >> GPLDA_CHECK;

  gplda::compute_d_idx(*stream, gpu_d_len, gpu_d_idx, n_docs);
  hipStreamSynchronize(*stream);

  hipMemcpy(d_idx, gpu_d_idx, size*sizeof(uint32_t), hipMemcpyDeviceToHost) >> GPLDA_CHECK;

  assert(d_idx[0] == 0);
  assert(d_idx[1] == 3);
  assert(d_idx[2] == 5);
  assert(d_idx[3] == 9);

  hipStreamDestroy(*stream);
  delete stream;

  hipFree(gpu_d_len);
  hipFree(gpu_d_idx);
}

void test_warp_sample_topics() {

}

}
