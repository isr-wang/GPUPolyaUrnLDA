#include "test_spalias.cuh"
#include "../spalias.cuh"
#include "../error.cuh"
#include "assert.h"

using gplda::FileLine;

namespace gplda_test {

void test_build_alias() {
  uint32_t table_size = 10;

  float** prob;
  float** alias;

  hipMalloc(&prob, sizeof(float*)) >> GPLDA_CHECK;
  hipMalloc(&alias, sizeof(float*)) >> GPLDA_CHECK;

  float** prob_host[1];
  float** alias_host[1];

  float prob_host_values[10] = {0.9,0.02,0.01,0.01,0.01, 0.01,0.01,0.01,0.01,0.01};
  float alias_host_values[10];

  hipMalloc(&prob_host[0], table_size * sizeof(float)) >> GPLDA_CHECK;
  hipMalloc(&alias_host[0], table_size * sizeof(float)) >> GPLDA_CHECK;

  hipMemcpy(prob, prob_host, sizeof(float*), hipMemcpyHostToDevice) >> GPLDA_CHECK;
  hipMemcpy(alias, alias_host, sizeof(float*), hipMemcpyHostToDevice) >> GPLDA_CHECK;

  hipMemcpy(prob_host[0], prob_host_values, table_size * sizeof(float), hipMemcpyHostToDevice) >> GPLDA_CHECK;

  gplda::build_alias<<<1,64>>>(prob, alias, 10);
  hipDeviceSynchronize() >> GPLDA_CHECK;

  hipMemcpy(prob_host_values, prob_host[0], table_size * sizeof(float), hipMemcpyDeviceToHost) >> GPLDA_CHECK;
  hipMemcpy(alias_host_values, alias_host[0], table_size * sizeof(float), hipMemcpyDeviceToHost) >> GPLDA_CHECK;

  for(int32_t i = 0; i < table_size; ++i) {
    assert(prob_host_values[i] <= 0.02f);
    assert(alias_host_values[i] == 0.0f);
  }

  hipFree(prob_host[0]) >> GPLDA_CHECK;
  hipFree(alias_host[0]) >> GPLDA_CHECK;

  hipFree(prob) >> GPLDA_CHECK;
  hipFree(alias) >> GPLDA_CHECK;
}

}
