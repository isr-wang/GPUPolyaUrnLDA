#include "test_spalias.cuh"
#include "../spalias.cuh"
#include "../error.cuh"
#include "assert.h"

using gplda::FileLine;

namespace gplda_test {

void test_build_alias() {
  uint32_t table_size = 10;

  float** prob;
  float** alias;

  hipMalloc(&prob, sizeof(float*)) >> GPLDA_CHECK;
  hipMalloc(&alias, sizeof(float*)) >> GPLDA_CHECK;

  float** prob_host = new float*[1];
  float** alias_host = new float*[1];

  float* prob_host_values = new float[table_size];
  float* alias_host_values = new float[table_size];
  for(int32_t i = 0; i < table_size; ++i) {
    prob_host_values[i] = 0.01f;
  }
  prob_host_values[0] = 0.9;
  prob_host_values[1] = 0.02;

  hipMalloc(&prob_host[0], table_size * sizeof(float)) >> GPLDA_CHECK;
  hipMalloc(&alias_host[0], table_size * sizeof(float)) >> GPLDA_CHECK;

  hipMemcpy(prob, prob_host, sizeof(float*), hipMemcpyHostToDevice) >> GPLDA_CHECK;
  hipMemcpy(alias, alias_host, sizeof(float*), hipMemcpyHostToDevice) >> GPLDA_CHECK;

  hipMemcpy(prob_host[0], prob_host_values, table_size * sizeof(float), hipMemcpyHostToDevice) >> GPLDA_CHECK;

  gplda::build_alias<<<1,64>>>(prob, alias, 10);
  hipDeviceSynchronize() >> GPLDA_CHECK;

  hipMemcpy(prob_host_values, prob_host[0], table_size * sizeof(float), hipMemcpyDeviceToHost) >> GPLDA_CHECK;
  hipMemcpy(alias_host_values, alias_host[0], table_size * sizeof(float), hipMemcpyDeviceToHost) >> GPLDA_CHECK;

  for(int32_t i = 0; i < table_size; ++i) {
    assert(prob_host_values[i] <= 1.0f);
    assert(alias_host_values[i] == 0.0f);
  }

  hipFree(prob_host[0]) >> GPLDA_CHECK;
  hipFree(alias_host[0]) >> GPLDA_CHECK;

  hipFree(prob) >> GPLDA_CHECK;
  hipFree(alias) >> GPLDA_CHECK;

  delete[] prob_host_values;
  delete[] alias_host_values;

  delete[] prob_host;
  delete[] alias_host;
}

}
