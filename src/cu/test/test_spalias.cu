#include "test_spalias.cuh"
#include "../spalias.cuh"
#include "../error.cuh"
#include "assert.h"

using gplda::FileLine;
using gplda::f32;
using gplda::i32;
using gplda::u32;
using gplda::u64;

namespace gplda_test {

void test_build_alias() {
  u32 table_size = 10;

  f32** prob;
  u32** alias;

  hipMalloc(&prob, sizeof(f32*)) >> GPLDA_CHECK;
  hipMalloc(&alias, sizeof(u32*)) >> GPLDA_CHECK;

  f32** prob_host[1];
  u32** alias_host[1];

  f32 prob_host_values[10] = {0.9,0.02,0.01,0.01,0.01, 0.01,0.01,0.01,0.01,0.01};
  u32 alias_host_values[10];

  hipMalloc(&prob_host[0], table_size * sizeof(f32)) >> GPLDA_CHECK;
  hipMalloc(&alias_host[0], table_size * sizeof(u32)) >> GPLDA_CHECK;

  hipMemcpy(prob, prob_host, sizeof(f32*), hipMemcpyHostToDevice) >> GPLDA_CHECK;
  hipMemcpy(alias, alias_host, sizeof(u32*), hipMemcpyHostToDevice) >> GPLDA_CHECK;

  hipMemcpy(prob_host[0], prob_host_values, table_size * sizeof(f32), hipMemcpyHostToDevice) >> GPLDA_CHECK;

  gplda::build_alias<<<1,64>>>(prob, alias, 10);
  hipDeviceSynchronize() >> GPLDA_CHECK;

  hipMemcpy(prob_host_values, prob_host[0], table_size * sizeof(f32), hipMemcpyDeviceToHost) >> GPLDA_CHECK;
  hipMemcpy(alias_host_values, alias_host[0], table_size * sizeof(u32), hipMemcpyDeviceToHost) >> GPLDA_CHECK;

  for(i32 i = 0; i < table_size; ++i) {
    assert(prob_host_values[i] <= 0.02f);
    assert(alias_host_values[i] == 0);
  }

  hipFree(prob_host[0]) >> GPLDA_CHECK;
  hipFree(alias_host[0]) >> GPLDA_CHECK;

  hipFree(prob) >> GPLDA_CHECK;
  hipFree(alias) >> GPLDA_CHECK;
}

}
